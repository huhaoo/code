#include<stdio.h>
#include<hip/hip_runtime.h>
#define i64 long long
#define m 1024
#define n 100000000
#define a(n) n*((i64)n*3-1)/2
/*
运行1/3的时间出97，出题人的恶意：
15431918 357216132022127  97
答案：
27042068 1096910149053902  108
*/
__global__ void search()
{
	int i=threadIdx.x+blockIdx.x*blockDim.x+1;
	int j=1,k=i-1,c=0; i64 I=a(i);
	while(j<i)
	{
		while(k>=j&&a(j)+a(k)>I) k--;
		if(k<j) break;
		if(a(j)+a(k)==I) c++;
		j++;
	}
	if(c>50){ printf("%d %lld  %d\n",i,a(i),c); }
}
int main()
{
	search<<<n/m+1,m>>>();
	hipDeviceSynchronize();
	printf("finished.\n");
	return 0;
}