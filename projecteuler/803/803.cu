#include<stdio.h>
#include<hip/hip_runtime.h>
#define i64 long long
#define u64 unsigned long long
#define f(x) ((x)*25214903917ull+11)&281474976710655ull
#define f20(x) ((x)*280364631195649ull+215216710221824ull)&281474976710655ull
#define st 144933752257087ull
__global__ void init(u64 *v)
{
	v[0]=st; for(int i=1;i<(1<<20)+10;i++) v[i]=f(v[i-1]);
//	printf("%llu %llu\n",v[0],v[123456]);
}
__global__ void search(u64 _id)
{
	u64 x=threadIdx.x+(blockIdx.x<<10)+(_id<<20),y=x;
//	printf("%llu\n",x);
	static const int l=9;
	static const int val[l]={41,20,25,25,11,4,40,13,4};
	for(int i=0;;i++)
	{
		if((x>>16)%52!=val[i]) break;
		if(i==l-1){ printf("%llu\n",y); break; }
		x=f(x);
	}
}
__global__ void update(u64 *v)
{
	u64 x=threadIdx.x+(blockIdx.x<<10);
	if(x<(1<<20)+10) v[x]=f20(v[x]);
}
__global__ void check(u64 _id,u64 *v)
{
	static const int l=9;
	static const int val[l]={37,20,2,10,24,45,4,23,19};
//	static const int val[l]={3,16,32,27,34,15,42,46,49};
	int x=threadIdx.x+(blockIdx.x<<10);
	for(int i=0;i<l;i++)
		if((v[i+x]>>16)%52!=val[i]) break;
		else if(i==l-1){ printf("%llu\n",x+(_id<<20)); }
}
int main()
{
/*	for(u64 i=0;i<(1<<28);i++)
	{
		search<<< (1<<10),(1<<10) >>>(i);
		cudaDeviceSynchronize();
//		if(i%(1<<8)==0) printf("%llu\n",i);
	}*/
/*	u64 a=st;
	for(int i=0;i<12345678;i++) a=f(a);
	for(int i=0;i<9;i++){ printf("%d,",int((a>>16)%52)); a=f(a); }*/
	u64 *val; hipMalloc(&val,sizeof(u64)*(1<<20)+10);
	init<<<1,1>>>(val);
//	cudaError_t error = cudaGetLastError();
//	printf("CUDA error: %s\n", cudaGetErrorString(error));
	for(u64 i=0;i<(1<<28);i++)
	{
		check<<<(1<<10),(1<<10)>>>(i,val);
		hipDeviceSynchronize();
		update<<<(1<<10)+1,(1<<10)>>>(val);
		hipDeviceSynchronize();
//		if(i%(1<<8)==0) printf("%llu\n",i);
	}
	hipFree(val);
	printf("finished.\n");
	return 0;
}